#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

extern "C"
{
	__global__ void testKernel(int* addr, unsigned short param1, char param2)
	{
		addr[0] = param1 + param2;
	}
}

char* muGetErrorString(hipError_t result);

void muEC(int position) //checks and outputs error position and error string
{
	hipError_t errcode = hipGetLastError();
	if(errcode==hipSuccess)
	{
		printf("No error at position %i\n", position);
		return;
	}
	printf("Error position: %i\nCode:%s\n", position, hipGetErrorString(errcode));
}

void muRC(int position, hipError_t result)
{
	if(result==0)
		printf("Success at %i\n", position);
	else
		printf("Error at %i:%s\n", position, muGetErrorString(result));
}

char* muGetErrorString(hipError_t result)
{
	switch(result)
	{
	case 0:		return "Success";
	case 1:		return "Invalid value";
	case 2:		return "Out of memory";
	case 3:		return "Not Initialized";
	case 4:		return "Deinitialized";

	case 100:	return "No device";
	case 101:	return "Invalid device";

	case 200:	return "Invalid image";
	case 201:	return "Invalid context";
	case 202:	return "Context already current";
	case 205:	return "Map failed";
	case 206:	return "Unmap failed";
	case 207:	return "Array is mapped";
	case 208:	return "Already mapped";
	case 209:	return "No binary for GPU";
	case 210:	return "Already acquired";
	case 211:	return "Not mapped";

	case 300:	return "Invalid source";
	case 301:	return "File not found";

	case 400:	return "Invalid handle";
	case 500:	return "Not found";
	case 600:	return "Not ready";

	case 700:	return "Launch failed";
	case 701:	return "Launch out of resources";
	case 702:	return "Launch timeout";
	case 703:	return "Launch incompatible texturing";

	case 999:	return "Unknown";
	};
	return "Unknown";
}


int main( int argc, char** argv) 
{
	if(argc<3)
	{
		puts("arguments: cubinname kernelname length tcount interval choice");
		puts("	length: number of 4-byte elements to allocate in memory");
		puts("	tcount: number of threads");
		puts("	interval: number of output items per group");
		puts("	choice: 0, all; 1, odd group only; 2, even group only; 3: none");
		return 0;
	}
	int length = 8;
	if(argc>=4)
	{
		length = atoi(argv[3]);
	}
	int tcount = 1;
	if(argc>=5)
	{
		tcount = atoi(argv[4]);
	}
	int* cpu_output=new int[length];
	int size = sizeof(int)*length;
	int interval = 1;
	if(argc>=6)
	{
		interval = atoi(argv[5]);
	}
	bool odd = true;
	bool even = true;
	if(argc>=7)
	{
		int choice = atoi(argv[6]);
		if(choice==1)
			even = false;
		else if(choice==2)
			odd = false;
		else if(choice==3)
		{
			even = false;
			odd = false;
		}
	}
	hipDeviceptr_t gpu_output;
	hipDevice_t device;
	hipCtx_t context;

	muRC(100, hipInit(0));
	muRC(95, hipDeviceGet(&device, 0));
	muRC(92, hipCtxCreate(&context, hipDeviceScheduleSpin, device));
	//muRC(91, hipCtxSetCacheConfig(hipFuncCachePreferShared));
	muRC(91, hipCtxSetCacheConfig(hipFuncCachePreferL1));
	muRC(90, hipMalloc(&gpu_output, size));

	hipEvent_t eStart, eStop;
	muRC(89, hipEventCreateWithFlags(&eStart, hipEventDefault));
	muRC(88, hipEventCreateWithFlags(&eStop, hipEventDefault));
	hipModule_t module;
	hipFunction_t kernel;
	hipError_t result = hipModuleLoad(&module, argv[1]);
	muRC(0 , result);
	result = hipModuleGetFunction(&kernel, module, argv[2]);
	muRC(1, result); 
	int param = 0x1010;
	muRC(2, cuParamSetSize(kernel, 20));
	muRC(3, cuParamSetv(kernel, 0, &gpu_output, 8));
	muRC(3, cuParamSetv(kernel, 16, &param, 4));
	muRC(4, cuFuncSetBlockShape(kernel, tcount,1,1));

	muRC(41, hipEventRecord(eStart,0) );
	muRC(5, cuLaunch(kernel));
	muRC(51, hipEventRecord(eStop,0) );

	muRC(6, hipMemcpyDtoH(cpu_output, gpu_output, size));
	muRC(7, hipCtxSynchronize());
	float time;
	muRC(75, hipEventElapsedTime(&time, eStart, eStop)); 
	printf("length=%i\n", length);
	printf("tcount=%i\n", tcount);
	printf("time=%f\n", time);
	for(int i=0; i<length/interval; i++)
	{
		if(i%2==0)
		{
			if(!even) continue;
		}
		else
		{
			if(!odd) continue;
		}
		for(int j=0; j<interval; j++)
			printf("i=%i, j=%i, output=%i\n", i, j, cpu_output[i*interval+j]);
		if(interval!=1)
			puts("");
	}
	muRC(8, hipModuleUnload(module));
	muRC(9, hipFree(gpu_output));
	muRC(10, hipCtxDestroy(context));
	delete[] cpu_output;
	return 0;
}
