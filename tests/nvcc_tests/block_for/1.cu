
#include <hip/hip_runtime.h>
__global__ void k(int *input, int *output, int count)
{
	int tid = threadIdx.x;
	int nid = blockDim.x;
	#pragma unroll 3
	for(int i =tid; i<count; i+=nid)
	{
		output[i]=input[i]*16;
	}
}


