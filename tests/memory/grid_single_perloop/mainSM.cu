#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

extern "C"
{
	__global__ void testKernel(int* addr, unsigned short param1, char param2)
	{
		addr[0] = param1 + param2;
	}
}

char* muGetErrorString(hipError_t result);

void muEC(int position) //checks and outputs error position and error string
{
	hipError_t errcode = hipGetLastError();
	if(errcode==hipSuccess)
	{
		printf("No error at position %i\n", position);
		return;
	}
	printf("Error position: %i\nCode:%s\n", position, hipGetErrorString(errcode));
}

void muRC(int position, hipError_t result)
{
	if(result==0)
		printf("Success at %i\n", position);
	else
		printf("Error at %i:%s\n", position, muGetErrorString(result));
}

char* muGetErrorString(hipError_t result)
{
	switch(result)
	{
	case 0:		return "Success";
	case 1:		return "Invalid value";
	case 2:		return "Out of memory";
	case 3:		return "Not Initialized";
	case 4:		return "Deinitialized";

	case 100:	return "No device";
	case 101:	return "Invalid device";

	case 200:	return "Invalid image";
	case 201:	return "Invalid context";
	case 202:	return "Context already current";
	case 205:	return "Map failed";
	case 206:	return "Unmap failed";
	case 207:	return "Array is mapped";
	case 208:	return "Already mapped";
	case 209:	return "No binary for GPU";
	case 210:	return "Already acquired";
	case 211:	return "Not mapped";

	case 300:	return "Invalid source";
	case 301:	return "File not found";

	case 400:	return "Invalid handle";
	case 500:	return "Not found";
	case 600:	return "Not ready";

	case 700:	return "Launch failed";
	case 701:	return "Launch out of resources";
	case 702:	return "Launch timeout";
	case 703:	return "Launch incompatible texturing";

	case 999:	return "Unknown";
	};
	return "Unknown";
}


int main( int argc, char** argv) 
{
	if(argc<3)
	{
		puts("arguments: cubinname kernelname length tcount interval choice");
		puts("	length: number of 4-byte elements to allocate in memory");
		puts("	tcount: number of threads");
		puts("	interval: number of output items per group");
		puts("	choice: 0, all; 1, odd group only; 2, even group only; 3: none");
		return 0;
	}
	int length = 8;
	if(argc>=4)
	{
		length = atoi(argv[3]);
	}
	int tcount = 1;
	if(argc>=5)
	{
		tcount = atoi(argv[4]);
	}
	int size = sizeof(int)*length;
	int interval = 1;
	if(argc>=6)
	{
		interval = atoi(argv[5]);
	}
	bool odd = true;
	bool even = true;
	if(argc>=7)
	{
		int choice = atoi(argv[6]);
		if(choice==1)
			even = false;
		else if(choice==2)
			odd = false;
		else if(choice==3)
		{
			even = false;
			odd = false;
		}
	}
	hipDeviceptr_t gpu_output;
	hipDevice_t device;
	hipCtx_t context;
	hipModule_t module;
	hipFunction_t kernel;
	hipEvent_t eStart, eStop;

	muRC(100, hipInit(0));
	muRC(95, hipDeviceGet(&device, 0));
	muRC(92, hipCtxCreate(&context, hipDeviceScheduleSpin, device));
	muRC(91, hipCtxSetCacheConfig(hipFuncCachePreferShared));
	//muRC(91, hipCtxSetCacheConfig(hipFuncCachePreferL1));
	muRC(90, hipMalloc(&gpu_output, size));

	muRC(89, hipEventCreateWithFlags(&eStart, hipEventDefault));
	muRC(88, hipEventCreateWithFlags(&eStop, hipEventDefault));
	muRC(0, hipModuleLoad(&module, argv[1]));
	muRC(1, hipModuleGetFunction(&kernel, module, argv[2]));

	muRC(2, cuParamSetSize(kernel, 20));
	muRC(3, cuParamSetv(kernel, 0, &gpu_output, 8));
	muRC(4, cuFuncSetBlockShape(kernel, tcount,1,1));

	muRC(41, hipEventRecord(eStart,0) );
	muRC(5, cuLaunchGrid(kernel, 150, 1));
	muRC(51, hipEventRecord(eStop,0) );

	muRC(7, hipCtxSynchronize());
	float time;
	muRC(75, hipEventElapsedTime(&time, eStart, eStop)); 

	printf("length=%i\n", length);
	printf("tcount=%i\n", tcount);
	printf("time=%f\n", time);
	printf("Allocation = %x, size = %x\n", gpu_output, size);
	
	muRC(8, hipModuleUnload(module));
	muRC(9, hipFree(gpu_output));
	muRC(10, hipCtxDestroy(context));
	return 0;
}
