
#include <hip/hip_runtime.h>
__global__ void k(int *input, int *output, int count)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int nid = blockDim.x*gridDim.x;
	#pragma unroll 3
	for(int i =tid; i<count; i+=nid)
	{
		output[i]=input[i]*16;
	}
}


