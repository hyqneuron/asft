#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

extern "C"
{
	__global__ void testKernel(int* addr, unsigned short param1, char param2)
	{
		addr[0] = param1 + param2;
	}
}

char* muGetErrorString(hipError_t result);
void muEC(int position); //checks and outputs error position and error string
void muRC(int position, hipError_t result);
char* muGetErrorString(hipError_t result);
bool ProcessCommandLine(char **argv, int argc, int &length, int &tcount, int &interval, int &choice,  bool &odd, bool &even);

int main( int argc, char** argv) 
{
	int length, tcount, interval, choice;
	bool odd, even;
	if(!ProcessCommandLine(argv, argc, length, tcount, interval, choice, odd, even))
		return -1;
	unsigned int cpu_loadee[] = {
		0x00005de4,
		0x28004404,
		0x94001c04,
		0x2c000000,
		0x84009c04,
		0x2c000000,
		0x8000dde4,
		0x28004000,
		0x20001ca3,
		0x20044000,
		0x10011ca3,
		0x2007c000,
		0x10009ce3,
		0x5000c000,
		0x90215c43,
		0x48004000,
		0x00401c85,
		0x94000000,
		0x00001de7,
		0x80000000};
	int loadee_size = sizeof(int) * 20;

//Initialize device and context, module and kernel func handles
	hipDeviceptr_t gpu_output, gpu_loadee, gpu_flag;
	hipDevice_t device;
	hipCtx_t context;
	hipModule_t module;
	hipFunction_t kernel;

	muRC(100, hipInit(0));
	muRC(95, hipDeviceGet(&device, 0));
	muRC(92, hipCtxCreate(&context, hipDeviceScheduleSpin, device));
	muRC(0, hipModuleLoad(&module, argv[1]));
	muRC(1, hipModuleGetFunction(&kernel, module, argv[2]));

//Prepare memory resources
	//tcount is used as block count, while a block uses 1024 threads
	bool addr_loaded = false;
	unsigned int container_addr = 0;
	length = tcount * 1024;
	int size = length * sizeof(int);
	int *cpu_output = new int[length];
	hipMalloc(&gpu_output, size);
	hipMalloc(&gpu_loadee, loadee_size);
	hipMalloc(&gpu_flag, 4);
	hipMemcpyHtoD(gpu_loadee, cpu_loadee, loadee_size);
	int flag = -1;
	hipMemcpyHtoD(gpu_flag, &flag, 4);


//set up events
	hipEvent_t eStart, eStop;
	muRC(89, hipEventCreateWithFlags(&eStart, hipEventDefault));
	muRC(88, hipEventCreateWithFlags(&eStop, hipEventDefault));

//set kernel parameters
LB1:
	if(addr_loaded)
	{
		//first launch flag = -1, signalling loader to output LEPC result
		//second launch flag = 0, signalling loader to load loadee
		flag = 0;
		hipMemcpyHtoD(gpu_flag, &flag, 4);
	}
	muRC(2, cuParamSetSize(kernel, 32));
	muRC(3, cuParamSetv(kernel, 0x0, &gpu_output, 8));
	muRC(3, cuParamSetv(kernel, 0x8, &gpu_loadee, 8));
	muRC(3, cuParamSetv(kernel, 0x10,&gpu_flag,   8));
	muRC(3, cuParamSetv(kernel, 0x18,&container_addr,4));
	muRC(3, cuParamSetv(kernel, 0x1c,&loadee_size,4));
	muRC(4, cuFuncSetBlockShape(kernel, 1024,1,1));

//launch, with events
	hipMemsetD32(gpu_output, 0, length);
	muRC(41, hipEventRecord(eStart,0) );
	muRC(5, cuLaunchGrid(kernel, tcount, 1)); //tcount used as block count
	muRC(51, hipEventRecord(eStop,0) );

//copy back result
	muRC(6, hipMemcpyDtoH(cpu_output, gpu_output, size));
	muRC(7, hipCtxSynchronize());
	float time;
	muRC(75, hipEventElapsedTime(&time, eStart, eStop)); 
	printf("length=%i\n", length);
	printf("tcount=%i\n", tcount);
	printf("time=%f\n", time);

	//first launch, getting PC and relaunch
	if(!addr_loaded)
	{
		addr_loaded = true;
		container_addr = cpu_output[0];
		printf("container addr is: %x\n", container_addr);
		container_addr += 0x400;
		printf("container addr to mod is: %x\n", container_addr);
		puts("=========Relaunching==========");
		goto LB1;
	}
	//second launch, check result
	else
	{
		bool error = false;
		for(int i = 0; i<length; i++)
		{
			if(cpu_output[i]!=i)
			{
				error = true;
				printf("error found when i = %x\n", i);
				break;
			}
		}
		if(error)
		{
			puts("error found");
			for(int i =0; i<length; i++)
				printf("i=%x, value = %x\n", i, cpu_output[i]);
		}
		else
			puts("kernel success");
	}

//clean up
	muRC(9, hipFree(gpu_output));
	muRC(9, hipFree(gpu_loadee));
	muRC(9, hipFree(gpu_flag));
	delete[] cpu_output;
	muRC(8, hipModuleUnload(module));
	muRC(10, hipCtxDestroy(context));
	return 0;
}


bool ProcessCommandLine(char **argv, int argc, int &length, int &tcount, int &interval, int &choice,  bool &odd, bool &even)
{
	if(argc<3)
	{
		puts("arguments: cubinname kernelname length tcount interval choice");
		puts("	length: number of 4-byte elements to allocate in memory");
		puts("	tcount: number of threads");
		puts("	interval: number of output items per group");
		puts("	choice: 0, all; 1, odd group only; 2, even group only; 3: none");
		return false;
	}
	length = 8;
	if(argc>=4)
	{
		length = atoi(argv[3]);
	}
	tcount = 1;
	if(argc>=5)
	{
		tcount = atoi(argv[4]);
	}
	interval = 1;
	if(argc>=6)
	{
		interval = atoi(argv[5]);
	}
	odd = true;
	even = true;
	if(argc>=7)
	{
		choice = atoi(argv[6]);
		if(choice==1)
			even = false;
		else if(choice==2)
			odd = false;
		else if(choice==3)
		{
			even = false;
			odd = false;
		}
	}
	return true;
}


void muEC(int position) //checks and outputs error position and error string
{
	hipError_t errcode = hipGetLastError();
	if(errcode==hipSuccess)
	{
		printf("No error at position %i\n", position);
		return;
	}
	printf("Error position: %i\nCode:%s\n", position, hipGetErrorString(errcode));
}

void muRC(int position, hipError_t result)
{
	if(result==0){}
		//printf("Success at %i\n", position);
	else
		printf("Error at %i:%s\n", position, muGetErrorString(result));
}

char* muGetErrorString(hipError_t result)
{
	switch(result)
	{
	case 0:		return "Success";
	case 1:		return "Invalid value";
	case 2:		return "Out of memory";
	case 3:		return "Not Initialized";
	case 4:		return "Deinitialized";

	case 100:	return "No device";
	case 101:	return "Invalid device";

	case 200:	return "Invalid image";
	case 201:	return "Invalid context";
	case 202:	return "Context already current";
	case 205:	return "Map failed";
	case 206:	return "Unmap failed";
	case 207:	return "Array is mapped";
	case 208:	return "Already mapped";
	case 209:	return "No binary for GPU";
	case 210:	return "Already acquired";
	case 211:	return "Not mapped";

	case 300:	return "Invalid source";
	case 301:	return "File not found";

	case 400:	return "Invalid handle";
	case 500:	return "Not found";
	case 600:	return "Not ready";

	case 700:	return "Launch failed";
	case 701:	return "Launch out of resources";
	case 702:	return "Launch timeout";
	case 703:	return "Launch incompatible texturing";

	case 999:	return "Unknown";
	};
	return "Unknown";
}
